#include "hip/hip_runtime.h"
#include <mergesort.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define BLOCKS_IN_GRID 380
#define THREADS_IN_BLOCK 256

#define min(a, b) (a < b ? a : b)

// bottom-up merge sort alg sources: https://en.wikipedia.org/wiki/Merge_sort#Bottom-up_implementation

__host__
void errorexit(const char *s) {
    printf("\n%s", s);	
    exit(EXIT_FAILURE);	 	
}

__device__ 
void merge_CUDA(int *arr, int *tmp, long left, long right, long end) {
    long i = left;
    long j = right;
    for (long k = left; k < end; k++) {
        if (i < right && (j >= end || arr[i] < arr[j])) {
            tmp[k] = arr[i];
            i++;
        } else {
            tmp[k] = arr[j];
            j++;
        }
    }
}

__global__ 
void mergeSortParallel_CUDA(int *arr, int *tmp, long size, long width, long chunksN) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long start = idx * width * chunksN;

    for (long i = 0; i < chunksN; i++) {
        if (start >= size)
            break;

        merge_CUDA(arr, tmp, start, min(start + (width >> 1), size), min(start + width, size));
        start += width;
    }
}

extern "C" void mergeSortCUDA(int *arr, long size) {
	
	clock_t begin, end;
	long threadsN = BLOCKS_IN_GRID * THREADS_IN_BLOCK;

	// device memory allocation (GPU)
    int *darr = NULL;
	int *dtmp = NULL;
    if (hipSuccess != hipMalloc((void **) &darr, size * sizeof(int))) {
		errorexit("Error allocating memory on the GPU");
    }
	if (hipSuccess != hipMalloc((void **) &dtmp, size * sizeof(int))) {
		errorexit("Error allocating memory on the GPU");
    }

	// copy arr to device memory
    if (hipSuccess != hipMemcpy(darr, arr, size * sizeof(int), hipMemcpyHostToDevice)) {
		errorexit("Error copying array to device");
	}

	// perform merge sort in bottom-up manner
	begin = clock();
	int* A = darr;
    int* B = dtmp;
	for (long width = 2; width < (size * 2); width *= 2) {
        long chunksN = size / ((threadsN) * width) + 1;

        mergeSortParallel_CUDA<<<BLOCKS_IN_GRID, THREADS_IN_BLOCK>>>(A, B, size, width, chunksN);
		hipMemcpy(A, B, size * sizeof(int), hipMemcpyDeviceToDevice);
    }
	end = clock();
	printf("| GPU actual calculations time: %fs\n", (double)(end - begin) / CLOCKS_PER_SEC);

	// copy results from device to host
    if (hipSuccess != hipMemcpy(arr, A, size * sizeof(int), hipMemcpyDeviceToHost)) {
		errorexit("Error copying results to host");
	}

	//free memory
    if (hipSuccess != hipFree(darr)) {
		errorexit("Error when deallocating space on the GPU");
	}
	if (hipSuccess != hipFree(dtmp)) {
		errorexit("Error when deallocating space on the GPU");
	}

    return;
}